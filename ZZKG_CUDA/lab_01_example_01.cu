#include "hip/hip_runtime.h"


#include <math.h>
#include <iostream>

const int N = 1024;
const int blocksize = 16;

__global__ void add_matrix(float *a, float *b, float *c, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j*N;
	if (i < N && j < N) {
		c[index] = a[index] + b[index];
	}
}

int main() {
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];
	for (int i = 0; i < N*N; ++i) {
		a[i] = 1.0f;
		b[i] = 3.5f; 
	}
	float *ad, *bd, *cd;
	const int size = N*N * sizeof(float);

	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMalloc((void**)&cd, size);

	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(ceil((float)N / (float)blocksize), ceil((float)N / blocksize));

	add_matrix <<< dimGrid, dimBlock >>>(ad, bd, cd, N);

	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

	for (int i = 0; i<10; i++) {
		std::cout << c[i] << "\n";
	}

	hipFree(ad); 
	hipFree(bd); 
	hipFree(cd);
	delete[] a; 
	delete[] b;
	delete[] c;

	return 0;
}